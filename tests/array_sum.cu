#include <cstddef>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>


__global__ void arraySum(const float* a, float *ret, int numElements) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < numElements) {
    __shared__ float sData[16];  // 申请共享数据内存
    sData[tid] = a[tid];
    __syncthreads();
    for (int i = numElements / 2; i > 0; i /= 2) {
      if (tid < i) {
        sData[tid] = sData[tid] + sData[tid + i];
      }
      __syncthreads();
    }
    if (tid == 0) {
      ret[0] = sData[0];
    }
  }
}

int main(int argc, char **argv) {

  // std::array<float, 16> a;
  // for (auto &i : a) {
  //   i *= (i + 1);
  // }

  int numElements = 16;
  size_t size = numElements * sizeof(float);

  float *h_data = (float *)malloc(size);;
  float *h_ret = (float *)malloc(size); // result on the host
  float* d_data = nullptr;  // data of device
  float *d_ret = nullptr; // result on the device

  for (int i = 0; i < numElements; ++ i) {
    h_data[i] = i * (i + 1);
  }

  hipMalloc((void**)&d_data, size);
  hipMalloc((void**)&d_ret, 1 * sizeof(float));

  hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

  arraySum<<<1, 16>>>(d_data, d_ret, numElements);
  hipMemcpy(h_ret, d_ret, 1 * sizeof(float), hipMemcpyDeviceToHost);

  // float temp[16];
  // cudaMemcpy(temp, d_data, 16 * sizeof(float), cudaMemcpyDeviceToHost);

  std::cout << "result: " << h_ret[0] << std::endl;
  hipFree(d_data);
  hipFree(d_ret);
  free(h_data);
  free(h_ret);
  return 0;
}
